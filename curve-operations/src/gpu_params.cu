#include "gpu_constants.cuh"
#include "gpu_fq.cuh"
#include "gpu_params.cuh"

using namespace std;

__constant__ HostParams host_params;
__device__ GpuParams gpu_params;

template <typename fixnum>
struct init_params_gpu
{
  __device__ void operator()(fixnum dummy)
  {
    // Converting all array's to fixnums and setting the parameters on the gpu
    gpu_params.set_mnt_mod(modnum(array_to_fixnum(host_params.get_mnt_mod())));
    gpu_params.set_mnt_non_residue(array_to_fixnum(host_params.get_mnt_non_residue()));
    gpu_params.set_mnt_coeff_a(array_to_fixnum(host_params.get_mnt_coeff_a()));
    gpu_params.set_mnt_coeff_a2(array_to_fixnum(host_params.get_mnt_coeff_a2_c0()), array_to_fixnum(host_params.get_mnt_coeff_a2_c1()));
    gpu_params.set_mnt_coeff_a3(array_to_fixnum(host_params.get_mnt_coeff_a3_c0()), array_to_fixnum(host_params.get_mnt_coeff_a3_c1()), array_to_fixnum(host_params.get_mnt_coeff_a3_c2()));
  }

  __device__ fixnum array_to_fixnum(fixnum *arr)
  {
    return arr[fixnum::layout::laneIdx()];
  }
};

void init_params(HostParams &params)
{
  cout << "Entered init_params" << endl;
  my_fixnum_array *dummy = my_fixnum_array::create(1);
  // Copy params (on host) to host_params (on device)
  hipMemcpyToSymbol(HIP_SYMBOL(host_params), &params, sizeof(HostParams));
  // Creates the cuda memory streams and synchronizations -- takes in dummy input
  my_fixnum_array::template map<init_params_gpu>(dummy);
  delete dummy;
}

__device__
    GpuParams &
    get_gpu_params()
{
  return gpu_params;
}