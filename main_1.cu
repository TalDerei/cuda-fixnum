#include "hip/hip_runtime.h"
// Stage 1: Quadratic Extension Arithmetic

#include <cstdio>
#include <cstring>
#include <cassert>
#include <vector>

#include "fixnum/warp_fixnum.cu"
#include "array/fixnum_array.h"
#include "functions/modexp.cu"
#include "functions/multi_modexp.cu"
#include "modnum/modnum_monty_redc.cu"
#include "modnum/modnum_monty_cios.cu"

const unsigned int bytes_per_elem = 128;
const unsigned int io_bytes_per_elem = 96;

// GPU modulus
// Declares device variable in constant memory, accessible from all threads, with lifetime of application
__constant__ uint8_t gpu_modulus[bytes_per_elem];

using namespace std;
using namespace cuFIXNUM;

template< typename fixnum >
struct mul_and_convert {
  // redc may be worth trying over cios
  typedef modnum_monty_cios<fixnum> modnum;
  __device__ void operator()(fixnum &r, fixnum a, fixnum b) {
      // Derefercing the device variable
      modnum mod = modnum(*(fixnum *)gpu_modulus);

      fixnum sm;
      fixnum am;
      fixnum bm;
      
      // to_modnum and from_modnum seem to be converting to and from montgomery form
      mod.to_modnum(am, a);
      mod.to_modnum(bm, b);

      mod.mul(sm, am, bm);

      fixnum s;
      mod.from_modnum(s, sm);
      r = s;
  }
};

template< int fn_bytes, typename fixnum_array >
void print_fixnum_array(fixnum_array* res, int nelts) {
    int lrl = fn_bytes*nelts;
    uint8_t local_results[lrl];
    int ret_nelts;
    for (int i = 0; i < lrl; i++) {
      local_results[i] = 0;
    }
    res->retrieve_all(local_results, fn_bytes*nelts, &ret_nelts);

    for (int i = 0; i < lrl; i++) {
      printf("%i ", local_results[i]);
    }
    printf("\n");
}

template< int fn_bytes, typename fixnum_array >
vector<uint8_t*> get_fixnum_array(fixnum_array* res, int nelts) {
    int lrl = fn_bytes*nelts;
    uint8_t local_results[lrl];
    int ret_nelts;
    for (int i = 0; i < lrl; i++) {
      local_results[i] = 0;
    }
    res->retrieve_all(local_results, fn_bytes*nelts, &ret_nelts);
    vector<uint8_t*> res_v;
    for (int n = 0; n < nelts; n++) {
      uint8_t* a = (uint8_t*)malloc(fn_bytes*sizeof(uint8_t));
      for (int i = 0; i < fn_bytes; i++) {
        a[i] = local_results[n*fn_bytes + i];
      }
      res_v.emplace_back(a);
    }
    return res_v;
}


template< int fn_bytes, typename word_fixnum, template <typename> class Func >
std::vector<uint8_t*> compute_product(std::vector<uint8_t*> a, std::vector<uint8_t*> b, uint8_t* input_m_base) {
    // Interface with the fixnum instruction set (precision modular arithmetic library that targets CUDA)
    // So we're defining a special array 'fixnum_array' that can use the instruction set on CUDA
    typedef warp_fixnum<fn_bytes, word_fixnum> fixnum;
    typedef fixnum_array<fixnum> fixnum_array;

    int nelts = a.size();

    cout << "nelt size is: " << nelts << endl;
    cout << "size of fn_bytes is: " << fn_bytes << endl;
    cout << "size of fn_bytes * nelts is: " << fn_bytes * nelts << endl;

    // Allocate memory to uint8_t array 'input_a' of size 128 x 1024 bytes
    uint8_t *input_a = new uint8_t[fn_bytes * nelts];
    for (int i = 0; i < fn_bytes * nelts; ++i) {
      input_a[i] = a[i/fn_bytes][i%fn_bytes];
    }

    // Allocate memory to uint8_t array 'input_b' of size 128 x 1024 bytes
    uint8_t *input_b = new uint8_t[fn_bytes * nelts];
    for (int i = 0; i < fn_bytes * nelts; ++i) {
      input_b[i] = b[i/fn_bytes][i%fn_bytes];
    }
    
    // Copies data to the given symbol on the device 
    hipMemcpyToSymbol(HIP_SYMBOL(gpu_modulus), input_m_base, bytes_per_elem);

    // Converting uint8_t * array into fixnum_array
    fixnum_array *res, *in_a, *in_b;
    in_a = fixnum_array::create(input_a, fn_bytes * nelts, fn_bytes);
    in_b = fixnum_array::create(input_b, fn_bytes * nelts, fn_bytes);
    res = fixnum_array::create(nelts);

    // Calling GPU functions on these arrays
    fixnum_array::template map<Func>(res, in_a, in_b);

    // Print the array
    // print_fixnum_array<fn_bytes, fixnum_array>(in_a, nelts);
    // print_fixnum_array<fn_bytes, fixnum_array>(in_b, nelts);
    // print_fixnum_array<fn_bytes, fixnum_array>(inM, nelts);
    // print_fixnum_array<fn_bytes, fixnum_array>(res, nelts);

    vector<uint8_t*> v_res = get_fixnum_array<fn_bytes, fixnum_array>(res, nelts);

    //TODO to do stage 1 field arithmetic, instead of a map, do a reduce

    delete in_a;
    delete in_b;
    delete res;
    delete[] input_a;
    delete[] input_b;
    return v_res;
}

uint8_t* read_mnt_fq(FILE* inputs) {
  // Allocate a buffer of memory of of size 128 bytes
  uint8_t* buf = (uint8_t*)calloc(bytes_per_elem, sizeof(uint8_t));
  // the input is montgomery representation x * 2^768 whereas cuda-fixnum expects x * 2^1024 so we shift over by (1024-768)/8 bytes
  fread((void*)(buf), io_bytes_per_elem*sizeof(uint8_t), 1, inputs);
  return buf;
}

void write_mnt_fq(uint8_t* fq, FILE* outputs) {
  fwrite((void *) fq, io_bytes_per_elem * sizeof(uint8_t), 1, outputs);
}

void print_array(uint8_t* a) {
  for (int j = 0; j < 128; j++) {
    printf("%x ", ((uint8_t*)(a))[j]);
  }
  printf("\n");
}

int main(int argc, char* argv[]) {
  setbuf(stdout, NULL);

  // Declare two uint8_t arrays of size 128 for each field respectively

  // mnt4_q
  uint8_t mnt4_modulus[bytes_per_elem] = {1,128,94,36,222,99,144,94,159,17,221,44,82,84,157,227,240,37,196,154,113,16,136,99,164,84,114,118,233,204,90,104,56,126,83,203,165,13,15,184,157,5,24,242,118,231,23,177,157,247,90,161,217,36,209,153,141,237,160,232,37,185,253,7,115,216,151,108,249,232,183,94,237,175,143,91,80,151,249,183,173,205,226,238,34,144,34,16,17,196,146,45,198,196,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
  // mnt6_q
  uint8_t mnt6_modulus[bytes_per_elem] = {1,0,0,64,226,118,7,217,79,58,161,15,23,153,160,78,151,87,0,63,188,129,195,214,164,58,153,52,118,249,223,185,54,38,33,41,148,202,235,62,155,169,89,200,40,92,108,178,157,247,90,161,217,36,209,153,141,237,160,232,37,185,253,7,115,216,151,108,249,232,183,94,237,175,143,91,80,151,249,183,173,205,226,238,34,144,34,16,17,196,146,45,198,196,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};

  // Inputs are represented as array of n uint64 ints in the field
  // The elements are represented using the Montgomery representation
  // x0, x1 : Array(𝔽MNT4753.q, n) and y0, y1 : Array(𝔽MNT6753.q, n)
  auto inputs = fopen(argv[2], "r"); 
  auto outputs = fopen(argv[3], "w");

  size_t n;

   while (true) {
    size_t elts_read = fread((void *) &n, sizeof(size_t), 1, inputs);
    cout << "objects read is: " << n << endl;
    if (elts_read == 0) { break; }

    // Initialize vectors of pointers to uint8_t and read input file into the vectors for each field
    std::vector<uint8_t*> x0;
    for (size_t i = 0; i < n; ++i) {
      x0.emplace_back(read_mnt_fq(inputs));
    }
    // Returns the number of elements held in the vector which is 1024 elements each 1 byte in size
    cout << "size of x0 is: " << x0.size() << endl;

    std::vector<uint8_t*> x1;
    for (size_t i = 0; i < n; ++i) {
      x1.emplace_back(read_mnt_fq(inputs));
    }
    cout << "size of x1 is: " << x1.size() << endl;

    // Call compute_product passing in the arrays and vectors for the mnt4_q field
    std::vector<uint8_t*> res_x = compute_product<bytes_per_elem, u64_fixnum, mul_and_convert>(x0, x1, mnt4_modulus);

    for (size_t i = 0; i < n; ++i) {
      write_mnt_fq(res_x[i], outputs);
    }

    std::vector<uint8_t*> y0;
    for (size_t i = 0; i < n; ++i) {
      y0.emplace_back(read_mnt_fq(inputs));
    }

    std::vector<uint8_t*> y1;
    for (size_t i = 0; i < n; ++i) {
      y1.emplace_back(read_mnt_fq(inputs));
    }

    // Call compute_product passing in the arrays and vectors for the mnt6_q field
    std::vector<uint8_t*> res_y = compute_product<bytes_per_elem, u64_fixnum, mul_and_convert>(y0, y1, mnt6_modulus);

    for (size_t i = 0; i < n; ++i) {
      write_mnt_fq(res_y[i], outputs);
    }

    for (size_t i = 0; i < n; ++i) {
      free(x0[i]);
      free(x1[i]);
      free(y0[i]);
      free(y1[i]);
      free(res_x[i]);
      free(res_y[i]);
    }
  }

  return 0;
}

