// Stage 2: Cubic Extension Arithmetic (3 fields)

#include <cstdio>
#include <cstring>
#include <cassert>
#include <vector>

#include "fixnum/warp_fixnum.cu"
#include "array/fixnum_array.h"
#include "functions/modexp.cu"
#include "functions/multi_modexp.cu"
#include "modnum/modnum_monty_redc.cu"
#include "modnum/modnum_monty_cios.cu"

using namespace std;
using namespace cuFIXNUM;

const unsigned int bytes_per_elem = 128;
const unsigned int io_bytes_per_elem = 96;

__constant__
const uint8_t non_residue_bytes[bytes_per_elem] = {11, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

__constant__
const uint8_t mnt6_modulus_bytes[bytes_per_elem] = {1,0,0,64,226,118,7,217,79,58,161,15,23,153,160,78,151,87,0,63,188,129,195,214,164,58,153,52,118,249,223,185,54,38,33,41,148,202,235,62,155,169,89,200,40,92,108,178,157,247,90,161,217,36,209,153,141,237,160,232,37,185,253,7,115,216,151,108,249,232,183,94,237,175,143,91,80,151,249,183,173,205,226,238,34,144,34,16,17,196,146,45,198,196,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};

// GPU modulus
// Declares device variable in constant memory, accessible from all threads, with lifetime of application
// __constant__ uint8_t gpu_modulus[bytes_per_elem];

// GpuFq class 
template <typename fixnum>
class GpuFq {
    // Instantiating modnum_monty_cios class -- interface for performing field arithmetic operations
    typedef modnum_monty_cios<fixnum> modnum;
private:
    fixnum data_;
    modnum& mod;

public:
    // Constructor 
    __device__ GpuFq(const fixnum &data, modnum &mod) : data_(data), mod(mod) {};

    // load function calls to_modnum() to perform z = x * R (mod) q in Monty form and returns GpuFq class instance 
    __device__ static GpuFq load(const fixnum &data, modnum &mod) {
        fixnum result;
        mod.to_modnum(result, data);
        return GpuFq(result, mod);
    }

    // loadMM just returns GpuFq class instance
    __device__ static GpuFq loadMM(const fixnum &data, modnum &mod) {
        return GpuFq(data, mod);
    }

    // save performs a multiplication and saves result in 'mod'
    __device__ __forceinline__ void save(fixnum &data) {
        this->mod.from_modnum(data, this->data_);
    }

    __device__ __forceinline__ void saveMM(fixnum &data) {
        data = this->data_;
    }

    // operator* performs a multiplication by multiplying two elements of type fixnum and returns GpuFq class instance 
    __device__ __forceinline__ GpuFq operator*(const GpuFq &other) const {
        fixnum result;
        this->mod.mul(result, this->data_, other.data_);
        return GpuFq(result, this->mod);
    }    

    // operator+ performs a addition by multiplying two elements of type fixnum and returns GpuFq class instance 
    __device__ __forceinline__ GpuFq operator+(const GpuFq &other) const {
      fixnum result;
      this->mod.add(result, this->data_, other.data_);
      return GpuFq(result, this->mod);
    }

    // operator- performs a subtraction by multiplying two elements of type fixnum and returns GpuFq class instance 
    __device__ __forceinline__ GpuFq operator-(const GpuFq &other) const {
      fixnum result;
      this->mod.sub(result, this->data_, other.data_);
      return GpuFq(result, this->mod);
    }

    __device__ __forceinline__ GpuFq squared() const {
      fixnum result;
      this->mod.sqr(result, this->data_);
      return GpuFq(result, this->mod);
    }

    __device__ __forceinline__ bool isZero() const {
      return fixnum::is_zero(this->data_);
    }
};

// GpuFq3 class 
template <typename fixnum>
class GpuFq3 {
    typedef GpuFq<fixnum> GpuFq;

private:
    GpuFq c0, c1, c2;
    GpuFq &non_residue;

public:
    __device__ GpuFq3(const GpuFq &c0, const GpuFq &c1, const GpuFq &c2, GpuFq &non_residue) : c0(c0), c1(c1), c2(c2), non_residue(non_residue) {}

    __device__ __forceinline__ void save(fixnum &c0, fixnum &c1, fixnum &c2) {
      this->c0.save(c0);
      this->c1.save(c1);
      this->c2.save(c2);
    }

    __device__ __forceinline__ void saveMM(fixnum &c0, fixnum &c1, fixnum &c2) {
      this->c0.saveMM(c0);
      this->c1.saveMM(c1);
      this->c2.saveMM(c2);
    }

    __device__ __forceinline__ GpuFq3 operator*(const GpuFq3 &other) const {
      GpuFq c0_c0 = this->c0 * other.c0;
      GpuFq c0_c1 = this->c0 * other.c1;
      GpuFq c0_c2 = this->c0 * other.c2;
      
      GpuFq c1_c0 = this->c1 * other.c0;
      GpuFq c1_c1 = this->c1 * other.c1;
      GpuFq c1_c2 = this->c1 * other.c2;
      
      GpuFq c2_c0 = this->c2 * other.c0;
      GpuFq c2_c1 = this->c2 * other.c1;
      GpuFq c2_c2 = this->c2 * other.c2;

      return GpuFq3(c0_c0 + this->non_residue * (c1_c2 + c2_c1), c0_c1 + c1_c0 + this->non_residue * c2_c2, c0_c2 + c1_c1 + c2_c0, this->non_residue);
    }

    __device__ __forceinline__ GpuFq3 operator+(const GpuFq3 &other) const {
        return GpuFq3(this->c0 + other.c0, this->c1 + other.c1, this->c2 + other.c2, this->non_residue);
    }
};

template<typename fixnum>
struct mul_and_convert {
  typedef modnum_monty_cios<fixnum> modnum;
  typedef GpuFq<fixnum> GpuFq;
  typedef GpuFq3<fixnum> GpuFq3;

  __device__ void operator()(fixnum &r0, fixnum &r1, fixnum &r2, fixnum a0, fixnum a1, fixnum a2, fixnum b0, fixnum b1, fixnum b2) {
    fixnum n = array_to_fixnum(non_residue_bytes);
    fixnum m = array_to_fixnum(mnt6_modulus_bytes);

    modnum mod = modnum(m);
    GpuFq non_residue = GpuFq::load(n, mod);
    GpuFq3 fqA = GpuFq3(GpuFq::load(a0, mod), GpuFq::load(a1, mod), GpuFq::load(a2, mod), non_residue);
    GpuFq3 fqB = GpuFq3(GpuFq::load(b0, mod), GpuFq::load(b1, mod), GpuFq::load(b2, mod), non_residue);
    GpuFq3 fqS = fqA * fqB;
    fqS.save(r0, r1, r2);
  }

  __device__ fixnum array_to_fixnum(const uint8_t* arr) {
    return fixnum(((fixnum*)arr)[fixnum::layout::laneIdx()]);
  }
};

template< int fn_bytes, typename fixnum_array >
void print_fixnum_array(fixnum_array* res, int nelts) {
    int lrl = fn_bytes*nelts;
    uint8_t local_results[lrl];
    int ret_nelts;
    for (int i = 0; i < lrl; i++) {
      local_results[i] = 0;
    }
    res->retrieve_all(local_results, fn_bytes*nelts, &ret_nelts);

    for (int i = 0; i < lrl; i++) {
      printf("%i ", local_results[i]);
    }
    printf("\n");
}

template< int fn_bytes, typename fixnum_array >
uint8_t* get_fixnum_array(fixnum_array* res0, fixnum_array* res1, fixnum_array* res2, int nelts) {
    int lrl = fn_bytes*nelts;
    uint8_t* local_results0 = new uint8_t[lrl]; 
    uint8_t* local_results1 = new uint8_t[lrl];
    uint8_t* local_results2 = new uint8_t[lrl];
    int ret_nelts;
    for (int i = 0; i < lrl; i++) {
      local_results0[i] = 0;
      local_results1[i] = 0;
      local_results2[i] = 0;
    }
    res0->retrieve_all(local_results0, fn_bytes*nelts, &ret_nelts);
    res1->retrieve_all(local_results1, fn_bytes*nelts, &ret_nelts);
    res2->retrieve_all(local_results2, fn_bytes*nelts, &ret_nelts);

    uint8_t* local_results = new uint8_t[3 * lrl]; 

    for (int i = 0; i < nelts; i++) {
      mempcpy(local_results + 3 * i * fn_bytes, local_results0 + i * fn_bytes, fn_bytes);
      mempcpy(local_results + 3 * i * fn_bytes + fn_bytes, local_results1 + i * fn_bytes, fn_bytes);
      mempcpy(local_results + 3 * i * fn_bytes + 2 * fn_bytes, local_results2 + i * fn_bytes, fn_bytes);
    }

    delete local_results0;
    delete local_results1;
    delete local_results2;
    return local_results;
}


template< int fn_bytes, typename word_fixnum, template <typename> class Func >
uint8_t* compute_product(uint8_t* a, uint8_t* b, int nelts) {
    // Interface with the fixnum instruction set (precision modular arithmetic library that targets CUDA)
    // So we're defining a special array 'fixnum_array' that can use the instruction set on CUDA
    typedef warp_fixnum<fn_bytes, word_fixnum> fixnum;
    typedef fixnum_array<fixnum> fixnum_array;

    // Allocate memory to uint8_t arrays of size 128 x 1024 bytes each
    cout << "fn_bytes * nelts is: " << fn_bytes * nelts << endl;
    uint8_t *input_a0 = new uint8_t[fn_bytes * nelts];
    uint8_t *input_a1 = new uint8_t[fn_bytes * nelts];
    uint8_t *input_a2 = new uint8_t[fn_bytes * nelts];

    for (int i = 0; i < nelts; i++) {
      mempcpy(input_a0 + i * fn_bytes, a + 3 * i * fn_bytes, fn_bytes);
      mempcpy(input_a1 + i * fn_bytes, a + 3 * i * fn_bytes + fn_bytes, fn_bytes);
      mempcpy(input_a2 + i * fn_bytes, a + 3 * i * fn_bytes + 2 * fn_bytes, fn_bytes);
    }

    uint8_t *input_b0 = new uint8_t[fn_bytes * nelts];
    uint8_t *input_b1 = new uint8_t[fn_bytes * nelts];
    uint8_t *input_b2 = new uint8_t[fn_bytes * nelts];

    for (int i = 0; i < nelts; i++) {
      mempcpy(input_b0 + i * fn_bytes, b + 3 * i * fn_bytes, fn_bytes);
      mempcpy(input_b1 + i * fn_bytes, b + 3 * i * fn_bytes + fn_bytes, fn_bytes);
      mempcpy(input_b2 + i * fn_bytes, b + 3 * i * fn_bytes + 2 * fn_bytes, fn_bytes);
    }

    // for (int i = 0; i < fn_bytes * nelts; ++i) {
    //   cout << unsigned(input_m[i]) << endl;
    // }

    fixnum_array *res0, *res1, *res2, *in_a0, *in_a1, *in_a2, *in_b0, *in_b1, *in_b2;

    // Converting uint8_t * array into fixnum_array
    // Create takes a pointer to the data, total bytes (fn_bytes * nelts), and bytes / element (fn_bytes)
    in_a0 = fixnum_array::create(input_a0, fn_bytes * nelts, fn_bytes);
    in_a1 = fixnum_array::create(input_a1, fn_bytes * nelts, fn_bytes);
    in_a2 = fixnum_array::create(input_a2, fn_bytes * nelts, fn_bytes);

    in_b0 = fixnum_array::create(input_b0, fn_bytes * nelts, fn_bytes);
    in_b1 = fixnum_array::create(input_b1, fn_bytes * nelts, fn_bytes);
    in_b2 = fixnum_array::create(input_b2, fn_bytes * nelts, fn_bytes);

    res0 = fixnum_array::create(nelts);
    res1 = fixnum_array::create(nelts);
    res2 = fixnum_array::create(nelts);

    // Calling 'mul_and_convert' struct that contains GPU functions to be performed on these arrays
    // This calls functions in fixnum_array.cu which allocate memory and synchronize memory between host and device
    fixnum_array::template map<Func>(res0, res1, res2, in_a0, in_a1, in_a2, in_b0, in_b1, in_b2);
    exit(0);

    uint8_t* v_res = get_fixnum_array<fn_bytes, fixnum_array>(res0, res1, res2, nelts);

    delete in_a0;
    delete in_a1;
    delete in_a2;
    delete in_b0;
    delete in_b1;
    delete in_b2;
    delete res0;
    delete res1;
    delete res2;
    delete[] input_a0;
    delete[] input_a1;
    delete[] input_a2;
    delete[] input_b0;
    delete[] input_b1;
    delete[] input_b2;
    return v_res;
    
    // Copies data to the given symbol on the device 
    // hipMemcpyToSymbol(HIP_SYMBOL(gpu_modulus), input_m_base, bytes_per_elem);

    // Print the array
}

void read_mnt_fq(uint8_t* dest, FILE* inputs) {
  // the input is montgomery representation x * 2^768 whereas cuda-fixnum expects x * 2^1024 so we shift over by (1024-768)/8 bytes
  fread((void*)(dest), io_bytes_per_elem*sizeof(uint8_t), 1, inputs);
}

void read_mnt_fq3(uint8_t* dest, FILE *inputs) {
  read_mnt_fq(dest, inputs);
  // the input is montgomery representation x * 2^768 whereas cuda-fixnum expects x * 2^1024 so we shift over by (1024-768)/8 bytes
  read_mnt_fq(dest + bytes_per_elem, inputs);
  read_mnt_fq(dest + 2 * bytes_per_elem, inputs);
}

void write_mnt_fq(uint8_t* src, FILE* outputs) {
  fwrite((void *) src, io_bytes_per_elem * sizeof(uint8_t), 1, outputs);
}

void write_mnt_fq3(uint8_t* src, FILE* outputs) {
  write_mnt_fq(src, outputs);
  // the input is montgomery representation x * 2^768 whereas cuda-fixnum expects x * 2^1024 so we shift over by (1024-768)/8 bytes
  write_mnt_fq(src + bytes_per_elem, outputs);
  write_mnt_fq(src + 2 * bytes_per_elem, outputs);
}

void print_array(uint8_t* a) {
  for (int j = 0; j < 128; j++) {
    printf("%x ", ((uint8_t*)(a))[j]);
  }
  printf("\n");
}

int main(int argc, char* argv[]) {
  // hipEvent_t start, stop;
  // hipEventCreate(&start);
  // hipEventCreate(&stop);
  // hipEventRecord(start);

  setbuf(stdout, NULL);

  auto inputs = fopen(argv[2], "r"); 
  auto outputs = fopen(argv[3], "w");

  size_t n;

   while (true) {
    size_t elts_read = fread((void *) &n, sizeof(size_t), 1, inputs);
    if (elts_read == 0) { break; }
    cout << "objects read is: " << n << endl;

    // Initialize pointer to dynamic uint8_t array -- size 2 * 128 * 1024 bytes
    uint8_t* x0 = new uint8_t[3 * n * bytes_per_elem];

    // Set every element to 0 in x0 
    memset(x0, 0, 3 * n * bytes_per_elem);

    for (size_t i = 0; i < n; ++i) {
      // The destination is shifting by starting position of x0 + 2 * i * bytes_per_elem (256) bytes.
      // This corresponds to 92 x 2 = 196 bytes + 64 byte shift (32 bytes for every element in the pair).
      // Each element in the finite field Fq2 is a pair (a0, a1)
      read_mnt_fq3(x0 + 3 * i * bytes_per_elem, inputs);
    }
    
    uint8_t* x1 = new uint8_t[3 * n * bytes_per_elem];
    memset(x1, 0, 3 * n * bytes_per_elem);
    for (size_t i = 0; i < n; ++i) {
      read_mnt_fq3(x1 + 3 * i * bytes_per_elem, inputs);
    }

    // Perform the multiplication
    uint8_t* res_x = compute_product<bytes_per_elem, u64_fixnum, mul_and_convert>(x0, x1, n);

    for (size_t i = 0; i < n; ++i) {
      write_mnt_fq3(res_x + 3 * i * bytes_per_elem, outputs);
    }

    delete[] x0;
    delete[] x1;
    delete[] res_x;

    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // float milliseconds = 0;
    // hipEventElapsedTime(&milliseconds, start, stop); 
    // cout << "time is: " << milliseconds << "ms" << endl;
  }

  return 0;
}

